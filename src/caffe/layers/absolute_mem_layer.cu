#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/absolute_mem_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void storeSign(const int n, const Dtype* bottom_data, char* bottom_sign_) {
  CUDA_KERNEL_LOOP(index, n) {
	  if (bottom_data[index]>0) {
		  bottom_sign_[index] = 1;
	  }
	  else if (bottom_data[index]<0) {
		  bottom_sign_[index] = -1;
	  }
	  else {
		  bottom_sign_[index] = 0;
	  }
  }
}

template <typename Dtype>
__global__ void backward_kernel(const int n, Dtype* bottom_diff, const char* bottom_sign_) {
  CUDA_KERNEL_LOOP(index, n) {
	  if (bottom_sign_[index]<0) {
		  bottom_diff[index] *= Dtype(-1);
	  }
	  else if (bottom_sign_[index]==0) {
		  bottom_diff[index] = 0;
	  }
  }
}

template <typename Dtype>
void AbsoluteMemLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
	const int count = bottom[0]->count();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const Dtype* bottom_data = bottom[0]->gpu_data();
	if (this->phase_==TEST) {
		caffe_gpu_abs(count, bottom[0]->gpu_data(), top_data);
	}
	else {
		storeSign<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, bottom_data, bottom_sign_.mutable_gpu_data());
		CUDA_POST_KERNEL_CHECK;
		caffe_gpu_abs(count, bottom[0]->gpu_data(), top_data);
	}
}

template<typename Dtype>
void AbsoluteMemLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
	const int count = top[0]->count();
	if (propagate_down[0]) {
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		backward_kernel<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, bottom_diff, bottom_sign_.gpu_data());
		CUDA_POST_KERNEL_CHECK;
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(AbsoluteMemLayer);


}  // namespace caffe
