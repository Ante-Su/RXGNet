#include "hip/hip_runtime.h"
#include "caffe/layers/fake_embed_jpeg_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void FakeEmbedJpegInterOnly(const int n, const int K, const int H,
		const int W, const Dtype* kernel_inter, const int size_inter,
		const Dtype* rand_location_inter, const Dtype* rand_operation,
		const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		const int w = index % W;
		const int h = index / W % H;
		const int k = index / W / H % K;
		const int n = index / W / H / K; // n-th stego
		const int index_cover = (((2 * n + 0) * K + k) * H + h) * W + w;
		const int index_stego = (((2 * n + 1) * K + k) * H + h) * W + w;
		const int diff = in[index_cover] - in[index_stego];

		const int siz_inter = (size_inter - 1) / 2;
		const int hstart_inter = max(h - siz_inter, 0);
		const int hend_inter = min(h + siz_inter, H);
		const int wstart_inter = max(w - siz_inter, 0);
		const int wend_inter = min(w + siz_inter, W);
		const int khstart_inter = hstart_inter - h + siz_inter;
		const int khend_inter = hend_inter - h + siz_inter;
		const int kwstart_inter = wstart_inter - w + siz_inter;
		const int kwend_inter = wend_inter - w + siz_inter;

		Dtype cumsum_inter = 0.;
		for (int i = khstart_inter; i < khend_inter; ++i) {
			for (int j = kwstart_inter; j < kwend_inter; ++j) {
				cumsum_inter += kernel_inter[i * size_inter + j];
			}
		}

		int fake_index_cover_inter = 0;
		int fake_index_stego_inter = 0;
		const float thres_inter = rand_location_inter[index] * cumsum_inter;
		cumsum_inter = 0;
		for (int i = khstart_inter; i < khend_inter; ++i) {
			for (int j = kwstart_inter; j < kwend_inter; ++j) {
				cumsum_inter += kernel_inter[i * size_inter + j];
				if (cumsum_inter >= thres_inter) {
					fake_index_cover_inter = (((2 * n + 0) * K + k) * H + h + i
							- siz_inter) * W + w + j - siz_inter;
					fake_index_stego_inter = (((2 * n + 1) * K + k) * H + h + i
							- siz_inter) * W + w + j - siz_inter;
					break;
				}
			}
		}

		if (rand_operation[index] > 0.5) {
			out[fake_index_stego_inter] = in[fake_index_cover_inter] + diff;
		} else {
			out[fake_index_stego_inter] = in[fake_index_cover_inter] - diff;
		}

	}
}

template<typename Dtype>
__global__ void FakeEmbedJpegInterIntra(const int n, const int K, const int H,
		const int W, const Dtype* kernel_inter, const Dtype* kernel_intra,
		const int size_inter, const int size_intra,
		const Dtype* rand_location_inter, const Dtype* rand_location_intra,
		const Dtype* rand_operation, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		const int w = index % W;
		const int h = index / W % H;
		const int k = index / W / H % K;
		const int n = index / W / H / K; // n-th stego
		const int index_cover = (((2 * n + 0) * K + k) * H + h) * W + w;
		const int index_stego = (((2 * n + 1) * K + k) * H + h) * W + w;
		const int diff = in[index_cover] - in[index_stego];

		const int siz_inter = (size_inter - 1) / 2;
		const int hstart_inter = max(h - siz_inter, 0);
		const int hend_inter = min(h + siz_inter, H);
		const int wstart_inter = max(w - siz_inter, 0);
		const int wend_inter = min(w + siz_inter, W);
		const int khstart_inter = hstart_inter - h + siz_inter;
		const int khend_inter = hend_inter - h + siz_inter;
		const int kwstart_inter = wstart_inter - w + siz_inter;
		const int kwend_inter = wend_inter - w + siz_inter;
		Dtype cumsum_inter = 0.;
		for (int i = khstart_inter; i < khend_inter; ++i) {
			for (int j = kwstart_inter; j < kwend_inter; ++j) {
				cumsum_inter += kernel_inter[i * size_inter + j];
			}
		}
		int fake_index_h_inter = 0;
		int fake_index_w_inter = 0;
		const float thres_inter = rand_location_inter[index] * cumsum_inter;
		cumsum_inter = 0;
		for (int i = khstart_inter; i < khend_inter; ++i) {
			for (int j = kwstart_inter; j < kwend_inter; ++j) {
				cumsum_inter += kernel_inter[i * size_inter + j];
				if (cumsum_inter >= thres_inter) {
					fake_index_h_inter =  h + i - siz_inter;
					fake_index_w_inter =  w + j - siz_inter;
					break;
				}
			}
		}


		const int dct_w = index % 8;
		const int dct_h = index / 8 % 8;

		const int siz_intra = (size_intra - 1) / 2;
		const int hstart_intra = max(dct_h - siz_intra, 0);
		const int hend_intra = min(dct_h + siz_intra, 8);
		const int wstart_intra = max(dct_w - siz_intra, 0);
		const int wend_intra = min(dct_w + siz_intra, 8);
		const int khstart_intra = hstart_intra - dct_h + siz_intra;
		const int khend_intra = hend_intra - dct_h + siz_intra;
		const int kwstart_intra = wstart_intra - dct_w + siz_intra;
		const int kwend_intra = wend_intra - dct_w + siz_intra;

		Dtype cumsum_intra = 0.;
		for (int i = khstart_intra; i < khend_intra; ++i) {
			for (int j = kwstart_intra; j < kwend_intra; ++j) {
				cumsum_intra += kernel_intra[i * size_intra + j];
			}
		}
		int fake_index_cover = 0;
		int fake_index_stego = 0;
		const float thres_intra = rand_location_intra[index] * cumsum_intra;
		cumsum_intra = 0;
		for (int i = khstart_intra; i < khend_intra; ++i) {
			for (int j = kwstart_intra; j < kwend_intra; ++j) {
				cumsum_intra += kernel_intra[i * size_intra + j];
				if (cumsum_intra >= thres_intra) {
					fake_index_cover = (((2 * n + 0) * K
							+ (dct_h + i - siz_intra) * 8 + dct_w + j
							- siz_intra) * H + fake_index_h_inter) * W
							+ fake_index_w_inter;
					fake_index_stego = (((2 * n + 1) * K
							+ (dct_h + i - siz_intra) * 8 + dct_w + j
							- siz_intra) * H + fake_index_h_inter) * W
							+ fake_index_w_inter;
					break;
				}
			}
		}


		if (rand_operation[index] > 0.5) {
			out[fake_index_stego] = in[fake_index_cover] + diff;
		} else {
			out[fake_index_stego] = in[fake_index_cover] - diff;
		}

	}
}

template<typename Dtype>
void FakeEmbedJpegLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
	if (this->phase_ == TRAIN) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = bottom[0]->mutable_gpu_data();
		const int count = bottom[0]->count() / 2;

		Blob<Dtype> rand_map_location_inter(num_ / 2, channels_, height_,
				width_);
		caffe_gpu_rng_uniform(rand_map_location_inter.count(), Dtype(0),
				Dtype(1), rand_map_location_inter.mutable_gpu_data());

		Blob<Dtype> rand_map_operation(num_ / 2, channels_, height_,
				width_);
		caffe_gpu_rng_uniform(rand_map_operation.count(), Dtype(0),
				Dtype(1), rand_map_operation.mutable_gpu_data());

		Blob<Dtype> temp(num_, channels_, height_, width_);

		for (int i = 0; i < num_; i += 2) {
			caffe_copy(count * 2 / num_, bottom_data + bottom[0]->offset(i),
					temp.mutable_gpu_data() + temp.offset(i));
			caffe_copy(count * 2 / num_, bottom_data + bottom[0]->offset(i),
					temp.mutable_gpu_data() + temp.offset(i + 1));
		}

		if (inter_only_) {
			FakeEmbedJpegInterOnly<Dtype> <<<CAFFE_GET_BLOCKS(count),
					CAFFE_CUDA_NUM_THREADS>>>(count, channels_, height_, width_,
					gaussian_kernel_inter_.gpu_data(), size_inter_,
					rand_map_location_inter.gpu_data(),
					rand_map_operation.gpu_data(), bottom_data,
					temp.mutable_gpu_data());
			CUDA_POST_KERNEL_CHECK;
		} else {
			Blob<Dtype> rand_map_location_intra(num_ / 2, channels_, height_,
					width_);
			caffe_gpu_rng_uniform(rand_map_location_intra.count(), Dtype(0),
					Dtype(1), rand_map_location_intra.mutable_gpu_data());

			FakeEmbedJpegInterIntra<Dtype> <<<CAFFE_GET_BLOCKS(count),
					CAFFE_CUDA_NUM_THREADS>>>(count, channels_, height_, width_,
					gaussian_kernel_inter_.gpu_data(),
					gaussian_kernel_intra_.gpu_data(), size_inter_, size_intra_,
					rand_map_location_inter.gpu_data(),
					rand_map_location_intra.gpu_data(),
					rand_map_operation.gpu_data(), bottom_data,
					temp.mutable_gpu_data());
			CUDA_POST_KERNEL_CHECK;
		}
		caffe_copy(temp.count(), temp.gpu_data(), top_data);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(FakeEmbedJpegLayer);

}  // namespace caffe
