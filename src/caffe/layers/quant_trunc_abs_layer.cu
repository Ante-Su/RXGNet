#include "hip/hip_runtime.h"
#include "caffe/layers/quant_trunc_abs_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void QuantTruncAbsForward(const int n, const int th, const Dtype* in,
		Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		if (in[index] >= 0) {
			out[index] = in[index];
		} else {
			out[index] = -in[index];
		}
		if ((out[index] - (int) out[index]) >= 0.5) {
			out[index] = (int) out[index] + 1;
		} else {
			out[index] = (int) out[index];
		}
		if (out[index] > th) {
			out[index] = th;
		}
	}
}

template<typename Dtype>
__global__ void QuantForward(const int n, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int sign = 0;
		if (in[index] >= 0) {
			sign = 1;
			out[index] = in[index];
		} else {
			sign = -1;
			out[index] = -in[index];
		}
		if ((out[index] - (int) out[index]) >= 0.5) {
			out[index] = (int) out[index] + 1;
		} else {
			out[index] = (int) out[index];
		}
		out[index] *= sign;
	}
}

template<typename Dtype>
__global__ void AbsForward(const int n, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		if (in[index] >= 0) {
			out[index] = in[index];
		} else {
			out[index] = -in[index];
		}
	}
}

template<typename Dtype>
__global__ void TruncForward(const int n, const int th, const Dtype* in,
		Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		if (in[index] > th) {
			out[index] = th;
		}
		if (in[index] < -th) {
			out[index] = -th;
		}
	}
}

template<typename Dtype>
__global__ void QuantTruncForward(const int n, const int th, const Dtype* in,
		Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int sign = 0;
		if (in[index] >= 0) {
			sign = 1;
			out[index] = in[index];
		} else {
			sign = -1;
			out[index] = -in[index];
		}
		if ((out[index] - (int) out[index]) >= 0.5) {
			out[index] = (int) out[index] + 1;
		} else {
			out[index] = (int) out[index];
		}
		if (out[index] > th) {
			out[index] = th;
		}
		out[index] *= sign;
	}
}

template<typename Dtype>
__global__ void QuantAbsForward(const int n, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		if (in[index] >= 0) {
			out[index] = in[index];
		} else {
			out[index] = -in[index];
		}
		if ((out[index] - (int) out[index]) >= 0.5) {
			out[index] = (int) out[index] + 1;
		} else {
			out[index] = (int) out[index];
		}
	}
}

template<typename Dtype>
__global__ void TruncAbsForward(const int n, const int th, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		if (in[index] >= 0) {
			out[index] = in[index];
		} else {
			out[index] = -in[index];
		}
		if (out[index] > th) {
			out[index] = th;
		}
	}
}

template<typename Dtype>
void QuantTruncAbsLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = top[0]->count();
	switch (this->layer_param_.quant_trunc_abs_param().process()) {
	case QuantTruncAbsParameter_ProcessMethod_QUANTTRUNCABS:
		QuantTruncAbsForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, threshold_, bottom_data,
				top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_QUANT:
		QuantForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, bottom_data, top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_ABS:
		AbsForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, bottom_data, top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_TRUNC:
		TruncForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, threshold_, bottom_data, top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_QUANTTRUNC:
		QuantTruncForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, threshold_, bottom_data,
				top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_QUANTABS:
		QuantAbsForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, top_data);
		break;
	case QuantTruncAbsParameter_ProcessMethod_TRUNCABS:
		TruncAbsForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, threshold_, bottom_data,
				top_data);
		break;
	default:
		LOG(FATAL)<< "Unknown process method.";
	}
	CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(QuantTruncAbsLayer);

}  // namespace caffe
