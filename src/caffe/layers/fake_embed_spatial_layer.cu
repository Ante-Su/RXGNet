#include "hip/hip_runtime.h"
#include "caffe/layers/fake_embed_spatial_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void FakeEmbedSpatial(const int n, const int K, const int H,
		const int W, const Dtype* kernel, const int size,
		const Dtype* rand_location, const Dtype* rand_operation,
		const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		const int w = index % W;
		const int h = index / W % H;
		const int k = index / W / H % K;
		const int n = index / W / H / K; // n-th stego
		const int index_cover = (((2 * n + 0) * K + k) * H + h) * W + w;
		const int index_stego = (((2 * n + 1) * K + k) * H + h) * W + w;
		const int diff = in[index_cover] - in[index_stego];

		const int siz = (size - 1) / 2;
		const int hstart = max(h - siz, 0);
		const int hend = min(h + siz, H);
		const int wstart = max(w - siz, 0);
		const int wend = min(w + siz, W);
		const int khstart = hstart - h + siz;
		const int khend = hend - h + siz;
		const int kwstart = wstart - w + siz;
		const int kwend = wend - w + siz;

		Dtype cumsum = 0.;
		for (int i = khstart; i < khend; ++i) {
			for (int j = kwstart; j < kwend; ++j) {
				cumsum += kernel[i * size + j];
			}
		}

		int fake_index_cover = 0;
		int fake_index_stego = 0;
		const float thres = rand_location[index] * cumsum;
		cumsum = 0;
		for (int i = khstart; i < khend; ++i) {
			for (int j = kwstart; j < kwend; ++j) {
				cumsum += kernel[i * size + j];
				if (cumsum >= thres) {
					fake_index_cover = (((2 * n + 0) * K + k) * H + h + i - siz)
							* W + w + j - siz;
					fake_index_stego = (((2 * n + 1) * K + k) * H + h + i - siz)
							* W + w + j - siz;
					break;
				}
			}
		}

		if (rand_operation[index] > 0.5) {
			out[fake_index_stego] = in[fake_index_cover] + diff;
		} else {
			out[fake_index_stego] = in[fake_index_cover] - diff;
		}

	}
}

template<typename Dtype>
void FakeEmbedSpatialLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
	if (this->phase_ == TRAIN) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = bottom[0]->mutable_gpu_data();
		const int count = bottom[0]->count() / 2;

		Blob<Dtype> rand_map_location(num_ / 2, channels_, height_, width_);
		caffe_gpu_rng_uniform(rand_map_location.count(), Dtype(0), Dtype(1),
				rand_map_location.mutable_gpu_data());

		Blob<Dtype> rand_map_operation(num_ / 2, channels_, height_, width_);
		caffe_gpu_rng_uniform(rand_map_operation.count(), Dtype(0), Dtype(1),
				rand_map_operation.mutable_gpu_data());

		Blob<Dtype> temp(num_, channels_, height_, width_);

		for (int i = 0; i < num_; i += 2) {
			caffe_copy(count * 2 / num_, bottom_data + bottom[0]->offset(i),
					temp.mutable_gpu_data() + temp.offset(i));
			caffe_copy(count * 2 / num_, bottom_data + bottom[0]->offset(i),
					temp.mutable_gpu_data() + temp.offset(i + 1));
		}

		FakeEmbedSpatial<Dtype> <<<CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS>>>(count, channels_, height_, width_,
				gaussian_kernel_.gpu_data(), size_,
				rand_map_location.gpu_data(), rand_map_operation.gpu_data(),
				bottom_data, temp.mutable_gpu_data());
		caffe_copy(temp.count(), temp.gpu_data(), top_data);

		CUDA_POST_KERNEL_CHECK;
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(FakeEmbedSpatialLayer);

}  // namespace caffe
