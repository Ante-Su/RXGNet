#include "hip/hip_runtime.h"
// Modified from implementation of ReLU layer
// The bn_conv layer relies on top_data for correct backward operations.
// But the following ReLU layer inevitably changes the top_data of BN output.
// This problem is solved by recovering the input data of the ReLU layer.

#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/relu_recover_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void ReLURecoverForward(const int n, const Dtype* in, Dtype* out,
		Dtype negative_slope) {
	CUDA_KERNEL_LOOP(index, n)
	{
		out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
	}
}

template<typename Dtype>
void ReLURecoverLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = bottom[0]->count();
	Dtype negative_slope = this->layer_param_.relu_param().negative_slope();

	if (this->phase_ == TRAIN) {
		if (bottom[0] == top[0]) {
			// For in-place computation
			caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
		}
	}

	ReLURecoverForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, bottom_data, top_data, negative_slope);
	CUDA_POST_KERNEL_CHECK
	;
}

template<typename Dtype>
__global__ void ReLURecoverBackward(const int n, const Dtype* in_diff,
		const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
	CUDA_KERNEL_LOOP(index, n)
	{
		out_diff[index] =
				in_diff[index]
						* ((in_data[index] > 0)
								+ (in_data[index] <= 0) * negative_slope);
	}
}

template<typename Dtype>
void ReLURecoverLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
	if (propagate_down[0]) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const int count = bottom[0]->count();
		Dtype negative_slope = this->layer_param_.relu_param().negative_slope();

		ReLURecoverBackward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, top_diff, bottom_data, bottom_diff, negative_slope);
		CUDA_POST_KERNEL_CHECK
		;

		if (this->phase_ == TRAIN) {
			if (bottom[0] == top[0]) {
				// For in-place computation
				caffe_copy(count, bottom_memory_.gpu_data(),
						bottom[0]->mutable_gpu_data());
			}
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(ReLURecoverLayer);

}  // namespace caffe
