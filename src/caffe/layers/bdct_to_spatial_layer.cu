#include "hip/hip_runtime.h"
#include "caffe/layers/bdct_to_spatial_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void Dequantize(const int n, const int H, const int W,
		const Dtype* quant, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int w = index % W;
		int h = index / W % H;
		int k = index / W / H % 64;
		int n = index / W / H / 64;
		out[(n * H * W + h * W + w) * 64 + k] = in[index] * quant[k];
	}
}

// The Idct function uses the code from: http://www.kurims.kyoto-u.ac.jp/~ooura/fft.html
// fft2d.zip (2006/12/28) file shrtdct.c
// Copyright Takuya OOURA, 1996-2001
template<typename Dtype>
__global__ void Idct(const int n, const Dtype* in, Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int offset = index * 64;

		Dtype C8_1R = 0.49039264020161522456;
		Dtype C8_1I = 0.09754516100806413392;
		Dtype C8_2R = 0.46193976625564337806;
		Dtype C8_2I = 0.19134171618254488586;
		Dtype C8_3R = 0.41573480615127261854;
		Dtype C8_3I = 0.27778511650980111237;
		Dtype C8_4R = 0.35355339059327376220;
		Dtype W8_4R = 0.70710678118654752440;

		Dtype x0r, x0i, x1r, x1i, x2r, x2i, x3r, x3i;
		Dtype xr, xi;

		for (int j = 0; j <= 7; j++) {
			x1r = C8_1R * in[offset + 1 * 8 + j]
					+ C8_1I * in[offset + 7 * 8 + j];
			x1i = C8_1R * in[offset + 7 * 8 + j]
					- C8_1I * in[offset + 1 * 8 + j];
			x3r = C8_3R * in[offset + 3 * 8 + j]
					+ C8_3I * in[offset + 5 * 8 + j];
			x3i = C8_3R * in[offset + 5 * 8 + j]
					- C8_3I * in[offset + 3 * 8 + j];
			xr = x1r - x3r;
			xi = x1i + x3i;
			x1r += x3r;
			x3i -= x1i;
			x1i = W8_4R * (xr + xi);
			x3r = W8_4R * (xr - xi);
			xr = C8_2R * in[offset + 2 * 8 + j]
					+ C8_2I * in[offset + 6 * 8 + j];
			xi = C8_2R * in[offset + 6 * 8 + j]
					- C8_2I * in[offset + 2 * 8 + j];
			x0r = C8_4R * (in[offset + 0 * 8 + j] + in[offset + 4 * 8 + j]);
			x0i = C8_4R * (in[offset + 0 * 8 + j] - in[offset + 4 * 8 + j]);
			x2r = x0r - xr;
			x2i = x0i - xi;
			x0r += xr;
			x0i += xi;
			out[offset + 0 * 8 + j] = x0r + x1r;
			out[offset + 7 * 8 + j] = x0r - x1r;
			out[offset + 2 * 8 + j] = x0i + x1i;
			out[offset + 5 * 8 + j] = x0i - x1i;
			out[offset + 4 * 8 + j] = x2r - x3i;
			out[offset + 3 * 8 + j] = x2r + x3i;
			out[offset + 6 * 8 + j] = x2i - x3r;
			out[offset + 1 * 8 + j] = x2i + x3r;
		}
		for (int j = 0; j <= 7; j++) {
			x1r = C8_1R * out[offset + j * 8 + 1]
					+ C8_1I * out[offset + j * 8 + 7];
			x1i = C8_1R * out[offset + j * 8 + 7]
					- C8_1I * out[offset + j * 8 + 1];
			x3r = C8_3R * out[offset + j * 8 + 3]
					+ C8_3I * out[offset + j * 8 + 5];
			x3i = C8_3R * out[offset + j * 8 + 5]
					- C8_3I * out[offset + j * 8 + 3];
			xr = x1r - x3r;
			xi = x1i + x3i;
			x1r += x3r;
			x3i -= x1i;
			x1i = W8_4R * (xr + xi);
			x3r = W8_4R * (xr - xi);
			xr = C8_2R * out[offset + j * 8 + 2]
					+ C8_2I * out[offset + j * 8 + 6];
			xi = C8_2R * out[offset + j * 8 + 6]
					- C8_2I * out[offset + j * 8 + 2];
			x0r = C8_4R * (out[offset + j * 8 + 0] + out[offset + j * 8 + 4]);
			x0i = C8_4R * (out[offset + j * 8 + 0] - out[offset + j * 8 + 4]);
			x2r = x0r - xr;
			x2i = x0i - xi;
			x0r += xr;
			x0i += xi;
			out[offset + j * 8 + 0] = x0r + x1r;
			out[offset + j * 8 + 7] = x0r - x1r;
			out[offset + j * 8 + 2] = x0i + x1i;
			out[offset + j * 8 + 5] = x0i - x1i;
			out[offset + j * 8 + 4] = x2r - x3i;
			out[offset + j * 8 + 3] = x2r + x3i;
			out[offset + j * 8 + 6] = x2i - x3r;
			out[offset + j * 8 + 1] = x2i + x3r;
		}
	}
}

template<typename Dtype>
__global__ void ModeToSpatial(const int n, const int H, const int W, const Dtype* in,
		Dtype* out) {
	CUDA_KERNEL_LOOP(index, n)
	{
		int dct_w = index % 8;
		int dct_h = index / 8 % 8;
		int blk_w = index / 64 % W;
		int blk_h = index / 64 / W % H;
		int n = index / 64 / W / H;
		out[n*W*H*64+(blk_h*8+dct_h)*W*8+blk_w*8+dct_w] = in[index];
	}
}

template<typename Dtype>
void BdctToSpatialLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = bottom[0]->count();
	const int count_block = num_ * height_ * width_;

	Blob<Dtype> dequantized(num_, 1, height_ * width_, 64);
	caffe_gpu_set(dequantized.count(), Dtype(0),
			dequantized.mutable_gpu_data());
	Blob<Dtype> idcted(num_, 1, height_ * width_, 64);
	caffe_gpu_set(idcted.count(), Dtype(0), idcted.mutable_gpu_data());

	Dequantize<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, height_, width_, quant_matrix_.gpu_data(), bottom_data,
			dequantized.mutable_gpu_data());
	CUDA_POST_KERNEL_CHECK;

	Idct<Dtype> <<<CAFFE_GET_BLOCKS(count_block), CAFFE_CUDA_NUM_THREADS>>>(
			count_block, dequantized.gpu_data(), idcted.mutable_gpu_data());
	CUDA_POST_KERNEL_CHECK;

	ModeToSpatial<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
			height_, width_, idcted.gpu_data(), top_data);
	CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(BdctToSpatialLayer);
}  // namespace caffe
